#include <stdio.h>
#include <assert.h>
#include <inttypes.h>
#include <string.h>
#include <hip/hip_runtime.h>
#define MAXN 1024
#define MaxProblem 1024
#define BLOCK_SIZE 512
 
#define UINT uint32_t
uint32_t hostMtx[2][MAXN*MAXN];
uint32_t Ret[2][MAXN*MAXN];
int problemindex=0;
int N;
//======================================================
__global__ void matrixAdd( int N,UINT* A, UINT* B, UINT* C){
    int r = blockIdx.x;
    int c = threadIdx.x;
    int ptr = r*N + c;
 
    C[ptr] = A[ptr] + B[ptr];
}
 
__global__ void matrixMul(  int N,UINT* A, UINT* B, UINT* C){
    int r = blockIdx.x;
    int c = threadIdx.x;
    int ptr = r*N + c;
 
    UINT sum = 0;
    for(int k=0; k<N; k++)
        sum += A[r*N + k] * B[k*N + c];
 
    C[ptr] = sum;
}
 
//============================================
 
void rand_gen(UINT c, int N, UINT A[MAXN*MAXN]) {
    UINT x = 2, n = N*N;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            x = (x * x + c + i + j)%n;
            A[i*N+j] = x;
        }
    }
}
 
UINT signature(int N, UINT A[MAXN*MAXN]) {
    UINT h = 0;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            h = (h + A[i*N+j]) * 2654435761LU;
    }
    return h;
}
//==========================================================
int main(int argc, char *argv[]) {
 
    uint32_t S[MaxProblem][64],TotalN[MaxProblem];
    while(scanf("%d", &TotalN[problemindex]) == 1){
 
    for (int i = 0; i < 2; i++) {
        scanf("%d", &S[problemindex][i]);
        }
        problemindex++;
    }
 
 
    //readIn();
    uint32_t *cuIN[2], *cuTmp[6];
    uint32_t memSz = MAXN*MAXN*sizeof(uint32_t);
    for (int i = 0; i < 2; i++) {
        hipMalloc((void **) &cuIN[i], memSz);
    }
    for (int i = 0; i < 6; i++)
        hipMalloc((void **) &cuTmp[i], memSz);
 
    for(int index=0;index<problemindex;index++){
 
        N=TotalN[index];
        #pragma omp parallel for
        for (int i = 0; i < 2; i++) {
            rand_gen(S[index][i], N,  hostMtx[i]);
            hipMemcpy(cuIN[i], hostMtx[i], memSz, hipMemcpyHostToDevice);
        }
        int num_blocks =  (N*N-1)/BLOCK_SIZE +1 ;
        // AB
        //multiply(cuIN[0], cuIN[1], cuTmp[0]);
        matrixMul<<<N, N>>>(N, cuIN[0], cuIN[1], cuTmp[0]);
        // BA
        //multiply(cuIN[1], cuIN[0], cuTmp[1]);
        matrixMul<<<N, N>>>(N, cuIN[1], cuIN[0], cuTmp[1]);
        //AB+BA
        //add(cuTmp[0], cuTmp[1], cuTmp[2]);
        matrixAdd<<<N, N>>>(N, cuTmp[0], cuTmp[1], cuTmp[2]);
 
        // ABA
        //multiply(cuTmp[0], cuIN[0], cuTmp[3]);
        matrixMul<<<N, N>>>(N, cuTmp[0], cuIN[0], cuTmp[3]);
        // BAB
        //multiply(cuTmp[1], cuIN[1], cuTmp[4]);
        matrixMul<<<N, N>>>(N, cuTmp[1], cuIN[1], cuTmp[4]);
        //ABA+BAB
        //add(cuTmp[3], cuTmp[4], cuTmp[5]);
        matrixAdd<<<N, N>>>(N, cuTmp[3], cuTmp[4], cuTmp[5]);
 
        hipMemcpy(Ret[0], cuTmp[2], memSz, hipMemcpyDeviceToHost);
        hipMemcpy(Ret[1], cuTmp[5], memSz, hipMemcpyDeviceToHost);
 
        uint32_t ret[2];
 
        #pragma omp parallel for
        for (int i = 0; i < 2; i++) {
            ret[i] = signature(N, Ret[i]);
        }
 
        for (int i = 0; i < 2; i++)
            printf("%u\n", ret[i]);
 
}
 
 
    for (int i = 0; i < 2; i++)
        hipFree(cuIN[i]);
    for (int i = 0; i < 6; i++)
        hipFree(cuTmp[i]);
    return 0;
}