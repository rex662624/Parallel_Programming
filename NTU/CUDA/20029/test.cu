#include <stdio.h>
#include <assert.h>
#include <inttypes.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#define MAXN 1024
#define MaxProblem 1024
#define BLOCK_SIZE 512
#define ThreadNumber 2 
#define UINT uint32_t

uint32_t hostMtx[ThreadNumber][2][MAXN*MAXN];
uint32_t Ret[ThreadNumber][2][MAXN*MAXN];
uint32_t ANS[MaxProblem][2];
int problemindex=0;
//======================================================
__global__ void matrixAdd( int N,UINT* A, UINT* B, UINT* C){
    int row = blockIdx.x;
    int col = threadIdx.x;
 
    C[row*N + col] = A[row*N + col] + B[row*N + col];
}
 
__global__ void matrixMul(  int N,UINT* A, UINT* B, UINT* C){
    int row = blockIdx.x;
    int col = threadIdx.x;
 
    UINT sum = 0;
    for(int k=0; k<N; k++)
        sum += A[row*N + k] * B[k*N + col];
 
    C[row*N + col] = sum;
}
 
//============================================
 
void rand_gen(UINT c, int N, UINT A[MAXN*MAXN]) {
    UINT x = 2, n = N*N;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            x = (x * x + c + i + j)%n;
            A[i*N+j] = x;
        }
    }
}
 
UINT signature(int N, UINT A[MAXN*MAXN]) {
    UINT h = 0;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++)
            h = (h + A[i*N+j]) * 2654435761LU;
    }
    return h;
}
//==========================================================
int main(int argc, char *argv[]) {

    omp_set_num_threads(ThreadNumber);
    uint32_t S[MaxProblem][64],TotalN[MaxProblem];
    while(scanf("%d", &TotalN[problemindex]) == 1){
 
    for (int i = 0; i < 2; i++) {
        scanf("%d", &S[problemindex][i]);
        }
        problemindex++;
    }
 
    
    //readIn();
    uint32_t *cuIN[ThreadNumber][2], *cuTmp[ThreadNumber][6];
    uint32_t memSz = MAXN*MAXN*sizeof(uint32_t);
    for(int k=0;k<ThreadNumber;k++){
        
        for (int i = 0; i < 2; i++) {
            hipSetDevice(k);
            hipMalloc((void **) &cuIN[k][i], memSz);
        }

        for (int i = 0; i < 6; i++){
            hipSetDevice(k);
            hipMalloc((void **) &cuTmp[k][i], memSz);
        }
    }

    #pragma omp parallel for schedule(dynamic , 1)
    for(int index=0;index<problemindex;index++){
        int pid = omp_get_thread_num();
        hipSetDevice(pid);
        int N=TotalN[index];

        for (int i = 0; i < 2; i++) {
            rand_gen(S[index][i], N,  hostMtx[pid][i]);
            hipMemcpy(cuIN[pid][i], hostMtx[pid][i], memSz, hipMemcpyHostToDevice);
        }
        // AB
        //multiply(cuIN[0], cuIN[1], cuTmp[0]);
        matrixMul<<<N, N>>>(N, cuIN[pid][0], cuIN[pid][1], cuTmp[pid][0]);
        // BA
        //multiply(cuIN[1], cuIN[0], cuTmp[1]);
        matrixMul<<<N, N>>>(N, cuIN[pid][1], cuIN[pid][0], cuTmp[pid][1]);
        //AB+BA
        //add(cuTmp[0], cuTmp[1], cuTmp[2]);
        matrixAdd<<<N, N>>>(N, cuTmp[pid][0], cuTmp[pid][1], cuTmp[pid][2]);
 
        // ABA
        //multiply(cuTmp[0], cuIN[0], cuTmp[3]);
        matrixMul<<<N, N>>>(N, cuTmp[pid][0], cuIN[pid][0], cuTmp[pid][3]);
        // BAB
        //multiply(cuTmp[1], cuIN[1], cuTmp[4]);
        matrixMul<<<N, N>>>(N, cuTmp[pid][1], cuIN[pid][1], cuTmp[pid][4]);
        //ABA+BAB
        //add(cuTmp[3], cuTmp[4], cuTmp[5]);
        matrixAdd<<<N, N>>>(N, cuTmp[pid][3], cuTmp[pid][4], cuTmp[pid][5]);
 
        hipMemcpy(Ret[pid][0], cuTmp[pid][2], memSz, hipMemcpyDeviceToHost);
        hipMemcpy(Ret[pid][1], cuTmp[pid][5], memSz, hipMemcpyDeviceToHost);

        for (int i = 0; i < 2; i++) {
            ANS[index][i] = signature(N, Ret[pid][i]);
        }
}
 
for(int index=0;index<problemindex;index++){
    for (int i = 0; i < 2; i++)
        printf("%u\n", ANS[index][i]);
    }

 for(int k =0;k<ThreadNumber;k++){
    for (int i = 0; i < 2; i++)
        hipFree(cuIN[k][i]);
    for (int i = 0; i < 6; i++)
        hipFree(cuTmp[k][i]);
    }
    return 0;
}