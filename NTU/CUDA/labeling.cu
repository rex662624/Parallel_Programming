#include "hip/hip_runtime.h"
#include "labeling.h"
#include "stdio.h"
#define BLOCK_SIZE 512
//global表示被host呼叫，但在device上執行。

__global__ void compute(const char *cuStr, int *cuPos, const int strLen) {
    //local shared memory in block
    //共有 BLOCK_SIZE 個 thread 在同一個 block 裡面
    __shared__ int local_pos[BLOCK_SIZE];

    //
    int global_index = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    int local_index = threadIdx.x;

    if (global_index >= strLen) {
        return;
    }

    // thrust::tabulate，將空白部分填入索引值，反之填入 -1
    local_pos[local_index] = (cuStr[global_index] != ' ') ? -1 : local_index;
    __syncthreads();

    int max = -1;
    // thrust::inclusive_scan，對初步得到的位置資訊 P0 運行 Prefix Maximum
    //先看自己前面1個 然後看2個，每一輪需要同步等其他人算完結果。
    for (int offset = 1; offset <= local_index; offset <<= 1) {
        if (local_pos[local_index] < local_pos[local_index-offset]) {
            local_pos[local_index] = local_pos[local_index-offset];
        }
        __syncthreads();
    }
    // thrust::tabulate, sub_offset
    cuPos[global_index] = local_index - local_pos[local_index];
}
//處理 連續的一段字母橫跨 2 個 block,因為block size 設512,而string最長500
__global__ void fix(int *cuPos, const int strLen) {
    int global_index = threadIdx.x + blockIdx.x*BLOCK_SIZE;
    int local_index = threadIdx.x;

    if (global_index >= strLen) {
        return;
    }

    // 檢查local第一個就是字母的，可能就有橫跨兩個block。(如果是一開始就是字母的字串，index等於字母數)
    if (blockIdx.x > 0 && cuPos[global_index] == (local_index+1)) {
        cuPos[global_index] += cuPos[(blockIdx.x*BLOCK_SIZE)-1];
    }
}

void labeling(const char *cuStr, int *cuPos, int strLen) {
    int num_blocks =  (strLen-1)/BLOCK_SIZE +1 ;
    // 函式名稱<<<block 數目(in a grid), thread 數目(in a block), shared memory 大小>>>(參數...);

    compute<<<num_blocks, BLOCK_SIZE>>>(cuStr, cuPos, strLen);
    fix<<<num_blocks, BLOCK_SIZE>>>(cuPos, strLen);
}